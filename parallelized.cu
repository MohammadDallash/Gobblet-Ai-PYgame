#include "hip/hip_runtime.h"
#include <cstring>
#include <ctime> 
#include <iostream>
#include <string>
#include <algorithm>
#include <cstdint>
#include <climits>
#include <random>

const int BOARD_SIZE = 4;
const int NUMBER_OF_PLAYERS = 2;
const int INVENTORY_SIZE = 3;
#define db(x) printf("%s\n", x)

using namespace std;

struct State
{
    // Integer called turn
    int turn;

    // 2D array of size BOARD_SIZE*BOARD_SIZE called board
    int board[BOARD_SIZE][BOARD_SIZE];

    // 2D array of size NUMBER_OF_PLAYERS*INVENTORY_SIZE
    int inventory[NUMBER_OF_PLAYERS][INVENTORY_SIZE];

    /* lastMove[0] = src, lastMove[1] = dest
     *
     * [0,1,1]
     * [0,2,3]
     *
    each of src and dest = [t, i, j]  (t is INVENTORY_MOVE OR BORAD_MOVE)*/
    int lastMove[2][3];
    int static_evl;
};



const int EMPTY_TILE = 0;
const int BLUE_SMALL = 1;
const int BLUE_MEDIUM = 2;
const int BLUE_LARGE = 4;
const int BLUE_XLARGE = 8;
const int ALL_BLUE = 15;

const int RED_SMALL = 16;
const int RED_MEDIUM = 32;
const int RED_LARGE = 64;
const int RED_XLARGE = 128;

#define fori(size) for (int i = 0; i < (size); i++)
#define forj(size) for (int j = 0; j < (size); j++)
#define fork(size) for (int k = 0; k < (size); k++)
const int INVENTORY_MOVE = 0;
const int BOARD_MOVE = 1;



__device__ int get_largest_piece(int n);
__device__ bool checkWins(State s);
__device__ int get_largest_piece_size(int n);
__device__ __host__ void debug_state(State state);
__device__ int static_evaluation(State curState);
__device__ bool customSort( State a,  State b);
__device__ void generate_possible_states(State curState, bool sorting,  int &n_child, State* &a);



using namespace std;

__device__ int get_largest_piece(int n)
{

    int pieces[] = {BLUE_XLARGE, RED_XLARGE,
                    BLUE_LARGE, RED_LARGE,
                    BLUE_MEDIUM, RED_MEDIUM,
                    BLUE_SMALL, RED_SMALL};

    for (int i = 0; i < 8; i++)
    {
        if (pieces[i] & n)
            return pieces[i];
    }

    return 0;
}


__device__ bool checkWins(State s) 
{
    int blue = 0;
    int red = 0;

    // Check rows
    for (int i = 0; i < 4; ++i) {
        for (int j = 0; j < 4; ++j) {
            if (get_largest_piece(s.board[i][j]) > ALL_BLUE && s.board[i][j] != EMPTY_TILE) {
                red++;
            } else if (get_largest_piece(s.board[i][j]) < RED_SMALL && s.board[i][j] != EMPTY_TILE) {
                blue++;
            }
        }

        if (red == 4) {
                   return true;

        } else if (blue == 4) {
                    return true;

        }

        // Reset counters
        blue = 0;
        red = 0;
    }

    // Check columns
    for (int i = 0; i < 4; ++i) {
        for (int j = 0; j < 4; ++j) {
            if (get_largest_piece(s.board[j][i]) > ALL_BLUE && s.board[j][i] != EMPTY_TILE) {
                red++;
            } else if (get_largest_piece(s.board[j][i]) < RED_SMALL && s.board[j][i] != EMPTY_TILE) {
                blue++;
            }
        }

        if (red == 4) {
            return true;
        } else if (blue == 4) {
            return true;
        }

        // Reset counters
        blue = 0;
        red = 0;
    }

    // Main diagonal
    for (int i = 0; i < 4; ++i) {
        if (get_largest_piece(s.board[i][i]) > ALL_BLUE && s.board[i][i] != EMPTY_TILE) {
            red++;
        } else if (get_largest_piece(s.board[i][i]) < RED_SMALL && s.board[i][i] != EMPTY_TILE) {
            blue++;
        }
    }

    if (red == 4) {
        return true;
    } else if (blue == 4) {
        return true;
    }

    // Reset counters
    blue = 0;
    red = 0;

    // Other diagonal
    for (int i = 0; i < 4; ++i) {
        if (get_largest_piece(s.board[i][3 - i]) > ALL_BLUE && s.board[i][3 - i] != EMPTY_TILE) {
            red++;
        } else if (get_largest_piece(s.board[i][3 - i]) < RED_SMALL && s.board[i][3 - i] != EMPTY_TILE) {
            blue++;
        }
    }

    if (red == 4) {
        return true;
    } else if (blue == 4) {
        return true;
    }

    return false;
}



__device__ int get_largest_piece_size(int n)
{

    int pieces[] = {BLUE_XLARGE, RED_XLARGE,
                    BLUE_LARGE, RED_LARGE,
                    BLUE_MEDIUM, RED_MEDIUM,
                    BLUE_SMALL, RED_SMALL};

    for (int i = 0; i < 8; i++)
    {
        if (pieces[i] & n)
            return 4 - i / 2; // return size only (color does NOT matter)
    }

    return 0;
}
__device__ __host__ void debug_state(State state) {
    printf("Turn: %d\n\n", state.turn);

    printf("Board:\n");
    fori(BOARD_SIZE) {
        forj(BOARD_SIZE) {
            printf("%d ", state.board[i][j]);
        }
        printf("\n");
    }

    printf("\nInventory:\n");
    fori(NUMBER_OF_PLAYERS) {
        forj(INVENTORY_SIZE) {
            printf("%d ", state.inventory[i][j]);
        }
        printf("\n");
    }
    printf("\n");

    printf("Last Move:\n\n");
    fori(2) {
        forj(INVENTORY_SIZE) {
            printf("%d ", state.lastMove[i][j]);
        }
        printf("\n");
    }
}


// blue is maximizer, red is minimizer
// the sign of the return value determines which is closer to winning
// the value determines how close to winning

// if the returned number is +ve then blue is closer to winning
// if the returned number is -ve then red is closer to winning
// the higher the positive number the closer is blue to winning
// the lower the negative number the closer is red to winning

//hueristics:
//1- number of red/blue pieces in each row/column/diagonal
//2- size of each piece in each row/column/diagonal


// unordered_map<unsigned long long, int> calculated_states;

__device__ int static_evaluation(State curState)
{
    // unsigned long long current_hash = computeHash(curState.board,curState.turn);
    // if(calculated_states.find(current_hash)!=calculated_states.end())
    // {
    //     return calculated_states[current_hash];
    // }

    // scores for each row, column, diagonal.
    int row[] = {0,0,0,0};
    int column[] = {0,0,0,0};
    int main_diagonal = 0;
    int other_diagonal = 0;

    int blue_won = 0;
    int red_won = 0;

    int blue_close = 0;
    int red_close = 0;

    

    // calculate the score of each row.
    for (int i = 0; i < 4; i++)
    {
        int blue = 0;
        int red = 0;

        //counters for blue and red without considering size
        int blue_count=0,red_count=0;

        for (int j = 0; j < 4; j++)
        {
            // if the piece is red and not an empty tile.
            if (get_largest_piece(curState.board[i][j]) > ALL_BLUE and curState.board[i][j] != EMPTY_TILE){
                red-=5; // its a red piece
                red_count--;
                red-=get_largest_piece_size(curState.board[i][j])*2; // also add its size
            }

            // if the piece is red blue and not an empty tile.
            if (get_largest_piece(curState.board[i][j]) < RED_SMALL and curState.board[i][j] != EMPTY_TILE){
                blue+=5; // its a blue piece
                blue_count++;
                blue+=get_largest_piece_size(curState.board[i][j])*2; // also add its size
            }
        }

        row[i] = blue + red;
        if(red_count == -3 && blue_count == 1)red_close += 10;
        if(blue_count == 3 && red_count == -1)blue_close += -10;
        if(red_count == -4)red_won = -1000;
        if(blue_count == 4)blue_won = 1000;

    }

    // columns
    for (int i = 0; i < 4; i++)
    {
        int blue = 0;
        int red = 0;
        int blue_count=0,red_count=0;
        for (int j = 0; j < 4; j++)
        {

            if (get_largest_piece(curState.board[j][i]) > ALL_BLUE and curState.board[j][i] != EMPTY_TILE){
                red-=5; // its a red piece
                red_count--;
                red-=get_largest_piece_size(curState.board[j][i])*2; // also add its size
            }

            if (get_largest_piece(curState.board[j][i]) < RED_SMALL and curState.board[j][i] != EMPTY_TILE){
                blue+=5; // its a blue piece
                blue_count++;
                blue+=get_largest_piece_size(curState.board[j][i])*2; // also add its size
            }
        }

        column[i] = blue + red;
        if(red_count == -3 && blue_count == 1)red_close += 10;
        if(blue_count == 3 && red_count == -1)blue_close += -10;
        if(red_count == -4)red_won = -1000;
        if(blue_count == 4)blue_won = 1000;
    }


    int blue = 0;
    int red = 0;
    int blue_count=0,red_count=0;
    // main diagonal
    for (int i = 0; i < 4; i++)
    {
        if (get_largest_piece(curState.board[i][i]) > 15 and curState.board[i][i] != 0){
            red-=5; // its a red piece
            red_count--;
            red-=get_largest_piece_size(curState.board[i][i])*2; // also add its size

        }

        if (get_largest_piece(curState.board[i][i]) < 16 and curState.board[i][i] != 0){
            blue+=5; // its a blue piece
            blue_count++;
            blue+=get_largest_piece_size(curState.board[i][i])*2; // also add its size
        }

    }


    main_diagonal = blue + red;
    if(red_count == -3 && blue_count == 1)red_close += 10;
    if(blue_count == 3 && red_count == -1)blue_close += -10;
    if(red_count == -4)red_won = -1000;
    if(blue_count == 4)blue_won = 1000;

    blue = 0;
    red = 0;
    blue_count = 0;
    red_count = 0;

    // other diagonal
    for (int i = 0; i < 4; i++)
    {
        if (get_largest_piece(curState.board[i][3 - i]) > ALL_BLUE and curState.board[i][3 - i] != EMPTY_TILE){
            red-=5; // its a red piece
            red_count--;
            red -= get_largest_piece_size(curState.board[i][3 - i])*2; // also add its size
        }

        if (get_largest_piece(curState.board[i][3 - i]) < RED_SMALL and curState.board[i][3 - i] != EMPTY_TILE){
            blue+=5; // its a blue piece
            blue_count++;
            blue += get_largest_piece_size(curState.board[i][3 - i])*2; // also add its size
        }

    }
    other_diagonal = blue + red;
    if(red_count == -3 && blue_count == 1)red_close += 10;
    if(blue_count == 3 && red_count == -1)blue_close += -10;
    if(red_count == -4)red_won = -1000;
    if(blue_count == 4)blue_won = 1000;

    // calculate the maximum - minimum
    int maxx = INT_MIN, minn = INT_MAX;

    fori(4)
    {
        maxx = max(row[i],maxx);
        minn = min(row[i],minn);

        maxx = max(column[i],maxx);
        minn = min(column[i],minn);
    }
    maxx = max(max(other_diagonal,main_diagonal),maxx);
    minn = min(min(other_diagonal,main_diagonal),minn);


    int result =10*(maxx + minn) + 3*(red_close + blue_close) + red_won + blue_won;

    // calculated_states[current_hash] = result;

    return  result;
}

__device__ bool customSort( State a,  State b)
{            
    return a.static_evl< b.static_evl;
        
}




__device__ int getFlattenedIndexInDst(int i, int j, int k)
{
    const int  d2 = 16, d3 = 3;

    return i * (d2 * d3) + j * d3 + k;
}


__device__ void generate_possible_states(State curState, bool sorting,  int &n_child, State* &a)
{

   

    if (checkWins(curState))
    {

        hipMalloc((void**)&a, sizeof(State) * 1);
        hipDeviceSynchronize();

        
        n_child ++;
        a[0] = curState;

        
        return;
    }


    
    const int d1 = 5, d2 = 16, d3 = 3;


    //  locations where each size exists (in the board and the inventory)

    int *possible_destination;
    hipMalloc((void**)&possible_destination, sizeof(int) * d1 * d2 * d3);
    hipDeviceSynchronize();



    int *p;
    hipMalloc((void**)&p, sizeof(int) * 5);

    hipDeviceSynchronize();

    p[0]=0;
    p[1]=0;
    p[2]=0;
    p[3]=0;
    p[4]=0;




    
    hipMalloc((void**)&a, sizeof(State) * 120);
    hipDeviceSynchronize();


    
    // add each location to its corresponding size

    fori(BOARD_SIZE)
    {
        forj(BOARD_SIZE)
        {
             
            int size = get_largest_piece_size(curState.board[i][j]);


            int idx = getFlattenedIndexInDst(size, p[size], 0);
            p[size]++;
            hipDeviceSynchronize();



            possible_destination[idx] = BOARD_MOVE;
            possible_destination[idx+1] = i;
            possible_destination[idx+2] = j;
        }
    }






    fori(BOARD_SIZE)
    {
        forj(BOARD_SIZE)
        {
            int curPiece = curState.board[i][j];

            int size = get_largest_piece_size(curPiece);

            int largest_piece = get_largest_piece(curPiece);



            if (((largest_piece > ALL_BLUE)) ^ (curState.turn))
                continue; // if its not your turn

            for (int s = 0; s < size; s++)
            {
                for (int d = 0; d < p[s]; d++)
                {
                    int *dest = possible_destination + getFlattenedIndexInDst(s,d,0);



                    State newState = curState;


                    newState.board[dest[1]][dest[2]] |= largest_piece;
                    newState.board[i][j] &= ~(largest_piece);



                    newState.lastMove[0][0] = BOARD_MOVE;
                    newState.lastMove[0][1] = i;
                    newState.lastMove[0][2]  = j;



                    newState.lastMove[1][0] = dest[0];
                    newState.lastMove[1][1] = dest[1];
                    newState.lastMove[1][2] = dest[2];




                    newState.turn = curState.turn ^ 1;
                    newState.static_evl=static_evaluation(newState);



                    a[n_child++] = newState;
                }
            }
        }
    }

  


    fori(INVENTORY_SIZE)
    {
        int curPiece = curState.inventory[curState.turn][i];

        int size = get_largest_piece_size(curPiece);

        int largest_piece = get_largest_piece(curPiece);
        
        


        for (int s = 0; s < size; s++)
        {
            for (int d = 0; d < p[s]; d++)
            {
                int *dest = possible_destination + getFlattenedIndexInDst(s,d,0);



                State newState = curState;


                newState.board[dest[1]][dest[2]] |= largest_piece;
                newState.inventory[curState.turn][i] &= ~(largest_piece);


                newState.lastMove[0][0] = INVENTORY_MOVE;
                newState.lastMove[0][1] = curState.turn;
                newState.lastMove[0][2]  = i;

                
                
                newState.lastMove[1][0] = dest[0];
                newState.lastMove[1][1] = dest[1];
                newState.lastMove[1][2] = dest[2];




                newState.turn = curState.turn ^ 1;
                newState.static_evl=static_evaluation(newState);



                
                a[n_child++] = newState;


            }
        }
    }


    hipFree(p);
    hipDeviceSynchronize();

    hipFree(possible_destination);
    hipDeviceSynchronize();


    if(sorting)
    {
        // sort(a, a + n_child, customSort);;
        ;
    }
}




__device__ State minMax_alpha_beta (State postion ,int depth,int alpha , int beta, bool buring, bool mutation, int difficulty)
{ 
    
    int evl;
    State temp;
    int n_child = 0;
    State * a;
    if(depth==0) return postion;

      

    generate_possible_states(postion, buring ,n_child ,a);


    printf(""); // magiccc

    hipDeviceSynchronize();










    if(postion.turn == 0)//maximizer
    {

        int largest_Eval=INT32_MIN;
        // reverse(a,a + n_child );
        for(int i=0;i<n_child;i++)
        {  
            State largest_state =minMax_alpha_beta (a[i], depth-1,alpha,beta, buring, mutation, difficulty);
            hipDeviceSynchronize();


            evl=largest_state.static_evl;
            alpha=max(evl,alpha);
            if(evl>largest_Eval or (evl== largest_Eval and mutation /*and rand()%3 == 1*/))
            {
                temp = a[i];
                largest_Eval = evl;
            }

            if(alpha>= beta and buring)break;
        }
    }
    else // minimizer
    {

        
        int minest_Eval=INT32_MAX;
        for(int i=0;i<n_child;i++)
        {
            State minest_state =minMax_alpha_beta(a[i], depth-1,alpha,beta, buring, mutation, difficulty);
            hipDeviceSynchronize();


            evl=minest_state.static_evl;
            beta=min(beta,evl);

            if(evl<minest_Eval or (evl== minest_Eval and mutation /*and rand()%3 == 1*/))
            {
                temp = a[i];
                minest_Eval = evl;
            }

            if(alpha>= beta and buring)  break;
            
        }
    }

    hipFree(a);
    hipDeviceSynchronize();


    return temp;
}

#pragma no_auto_parallel
__device__ State tt(State s)
{
    auto o = minMax_alpha_beta(s, 2, INT32_MIN, INT32_MAX, true, true,1 );
    hipDeviceSynchronize();

    return o;
}


__global__ void kernel(State s, State* o)
{
    State an = tt(s);
    *o = an;
}





int main(int argc, char *argv[]) 
{
    // fill_table();

    srand(static_cast<unsigned int>(time(0)));
    State initial_state;

    // Input the turn.
    initial_state.turn = atoi(argv[1]);

    // Input the board.
    int arg_index = 2;
    fori(BOARD_SIZE) {
        forj(BOARD_SIZE) {
            initial_state.board[i][j] = atoi(argv[arg_index++]);
        }
    }

    // Input the inventory.
    fori(NUMBER_OF_PLAYERS) {
        forj(INVENTORY_SIZE) {
            initial_state.inventory[i][j] = atoi(argv[arg_index++]);
        }
    }


    int  difficulty = atoi(argv[arg_index]);

    
    
    State *an;

    // Allocate memory for each vector on GPU
    hipMalloc(&an, sizeof(State));

    kernel<<<1, 1>>>(initial_state, an);
    hipDeviceSynchronize();

    // Allocate memory for anH on the host
    State *anH = (State*)malloc(sizeof(State));

    hipMemcpy(anH, an, sizeof(State), hipMemcpyDeviceToHost);

    // Now you can use anH as needed

    debug_state(*anH);


    // Don't forget to free the allocated memory on the host
    free(anH);

    // Don't forget to free the allocated memory on the device
    hipFree(an);
    return 0;
}

/*
./a.out 1 0 128 128 128  0 0 0 0  0 0 0 0  0 0 0 0   7 7 7  112 112 112  3
*/